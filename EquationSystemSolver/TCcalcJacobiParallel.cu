//============================================================================
// Name        : TCcalcJacobiParallel.cpp
// Author      : Niklas Bergh
//============================================================================


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <unordered_map>
#include <vector>
#include <string> // std::stoi
#include <algorithm>
#include <assert.h> // cudaCheckReturn

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif

#ifndef MAX_ITERATIONS
#define MAX_ITERATIONS 50
#endif

/* This solver uses the Jacobi/Gauss Seidel method, see https://www3.nd.edu/~zxu2/acms40390F12/Lec-7.3.pdf. It
 * converges for some systems, but not all
 */

__device__ int errorOnGPU;

__device__ inline int getValFromMatrix(int* matrix, int row, int col,int matSize,int pitchLength) {
	if (row<matSize && col<matSize) {return matrix[row*pitchLength + col];}
	return 0;
}

__device__ inline int getValFromVector(int* vector, int row, int matSize) {
	if (row<matSize) {return vector[row];}
	return 0;
}

__global__ void jacobiIterateKernel(int* cOnGPU,int* bOnGPU, int* xOnGPU, int* deltaXonGPU, int matSize, int pitchLength) {
	__shared__ int cShared[BLOCK_SIZE][BLOCK_SIZE+1]; // Make the columns BLOCK_SIZE + 1, so that there is less chance of a shared memory bank conflict
	__shared__ int xShared[BLOCK_SIZE];

	int myRow = blockIdx.x * blockDim.x + threadIdx.x;
	int myRowInBlock = threadIdx.x, myColInBlock = threadIdx.y;
	int rowSum=getValFromVector(bOnGPU,myRow,matSize);

	for (int m = 0; m < (matSize + BLOCK_SIZE - 1) / BLOCK_SIZE; m++) {
		cShared[myRowInBlock][myColInBlock] = getValFromMatrix(cOnGPU,myRow,m*BLOCK_SIZE+myColInBlock,matSize,pitchLength);
		if (myColInBlock==0) {xShared[myRowInBlock] = getValFromVector(xOnGPU,m*BLOCK_SIZE+myRowInBlock,matSize);}

		__syncthreads(); // Sync threads to make sure all fields have been written by all threads in the block to cShared and xShared

		if (myColInBlock==0) {
			for (int k=0;k<BLOCK_SIZE;k++) {
				// Only the Jacobi (not Gauss Seidel) iteration works in parallell:
				if (m*BLOCK_SIZE+k==myRow) {continue;}
				rowSum += cShared[myRowInBlock][k] * xShared[k];
			}
		}

		__syncthreads(); // Sync here so that no threads start changing the shared arrays (in the next iteration of m) before rowSum has been updated
	}

	if (myColInBlock==0 && myRow<matSize) {
		deltaXonGPU[myRow] = abs(rowSum - xOnGPU[myRow]);
		xOnGPU[myRow] = rowSum; // Update x
	}
}

__global__ void calculateErrorKernel(int* deltaXonGPU, int matSize) {
	__shared__ int deltaXshared[BLOCK_SIZE];
	__shared__ int sharedError;

	int myIndexInBlock = threadIdx.x;
	sharedError=0;

	for (int m = 0; m < (matSize + BLOCK_SIZE - 1) / BLOCK_SIZE;m++) {
		deltaXshared[myIndexInBlock] = getValFromVector(deltaXonGPU,m*BLOCK_SIZE+myIndexInBlock,matSize);

		__syncthreads();

		if (myIndexInBlock==0) {
			for (int k=0;k<BLOCK_SIZE;k++) {
				sharedError+=deltaXshared[k];
			}
		}

		__syncthreads();
	}
	if (myIndexInBlock==0) {
		errorOnGPU = sharedError;
	}
}

static inline void cudaCheckReturn(hipError_t result) {
	if (result != hipSuccess) {
		std::cerr <<"CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
		assert(result == hipSuccess);
	}
}

static int jacobiIterate(int* cOnGPU,int* bOnGPU, int* xOnGPU, int* deltaXonGPU, int matSize, int pitchLength) {
	int nrOfBlocks = (matSize + BLOCK_SIZE -1) / BLOCK_SIZE;
	dim3 threadSize(BLOCK_SIZE,BLOCK_SIZE);

	jacobiIterateKernel<<<nrOfBlocks,threadSize>>>(cOnGPU,bOnGPU,xOnGPU,deltaXonGPU,matSize,pitchLength);
	calculateErrorKernel<<<1,BLOCK_SIZE>>>(deltaXonGPU,matSize); // Alternatively deltaXonGPU can be copied to an array defined here, and iterate through it in CPU to get the error

	int error=0;
	// This call synchronizes the device with the host, so there is no application code running on the device past this point
	cudaCheckReturn(hipMemcpyFromSymbol(&error,HIP_SYMBOL(errorOnGPU),sizeof(int),0,hipMemcpyDeviceToHost));

	return error;
}

int main(int argc, char** argv) {
	if (argc<2) {std::cerr << "No equation file provided in command line argument" << std::endl; return -1;}

	// Check BLOCK_SIZE
	int blockSizeIn = BLOCK_SIZE;
	if (blockSizeIn<0 || blockSizeIn-BLOCK_SIZE!=0 || (blockSizeIn*blockSizeIn)%32!=0) {
		std::cerr << "Block size must be > 0, an integer and its square must be a multiple of 32" << std::endl;
		return -1;
	}

	// Check MAX_ITERATIONS
	int maxIterations = MAX_ITERATIONS;
	if (maxIterations<0 || maxIterations-MAX_ITERATIONS != 0) {
		std::cerr << "Illegal format of MAX_ITERATIONS" << std::endl;
		return -1;
	}

	// Start by reading the input file
	std::unordered_map<std::string, int> variableMap;
	std::vector<std::string> variableList;
	std::ifstream inFile;
	std::istringstream ss;
	std::string line,token;

	int** C,* b; // variable coefficients, equation constants
	int* x; // variables, variables in the new iteration

	int lineIndex=0,nrOfEquations=0,iters=0;
	bool firstToken=true;

	inFile.open(argv[1]);
	if (inFile.is_open()) {
		while (getline(inFile,line)) {
			ss.str(line);
			ss.clear();
			ss >> token;
			variableMap.insert({token,nrOfEquations});
			variableList.push_back(token);
			nrOfEquations++;
		}
		if (nrOfEquations==0) {std::cerr << "No equations in input file" << std::endl;return-1;}

		// Allocate the matrices, zero initialized with ()
		C = new int*[nrOfEquations];
		C[0] = new int[nrOfEquations*nrOfEquations];
		for (int i = 1; i < nrOfEquations;i++) {C[i] = &C[0][i*nrOfEquations];}
		b = new int[nrOfEquations]();
		x = new int[nrOfEquations]();

		inFile.clear();
		inFile.seekg(std::ios::beg); // Reset file pointer

		while (getline(inFile,line)) {
			C[lineIndex][lineIndex]=-1;
			ss.str(line);
			ss.clear();
			while (ss >> token) {
				if (firstToken) {firstToken=false;continue;}

				if (isalpha(token[0])) {
					C[lineIndex][variableMap.at(token)]++; // Add to coefficients
				}
				else if (isdigit(token[0])) {
					b[lineIndex]+=std::stoi(token); // Assuming no int overflow here
				}
			}
			if (C[lineIndex][lineIndex]!=-1) {
				/* C[lineIndex][lineIndex] cannot be zero. It is ok for it to be -1 or greater than 0. In the case it is not -1 or 0
				 * then all the other coefficients and b[lineIndex] needs to be divided by that -coefficient. I assume that it never happens
				 * here though, and only allow C[lineIndex][lineIndex] to be -1
				 */
				std::cerr << "Error, coefficient for diagonal variable is not 1" << std::endl;
				return -1;
			}
			firstToken=true;
			lineIndex++;
		}
	}
	else {
		std::cerr << "Unable to open file" << std::endl;
		return -1;
	}

	inFile.close();

	// Allocate data on the GPU:
	int* cOnGPU,* bOnGPU;
	int* xOnGPU,* deltaXonGPU;

	int rowSizeOnGPU = nrOfEquations * sizeof(int),pitchLength;
	size_t cPitch;

	cudaCheckReturn(hipMallocPitch(&cOnGPU,&cPitch,rowSizeOnGPU,nrOfEquations));
	cudaCheckReturn(hipMalloc(&bOnGPU,rowSizeOnGPU));
	cudaCheckReturn(hipMalloc(&xOnGPU,rowSizeOnGPU));
	cudaCheckReturn(hipMalloc(&deltaXonGPU,rowSizeOnGPU));

	pitchLength = cPitch/sizeof(int);

	cudaCheckReturn(hipMemcpy2D(cOnGPU,cPitch,C[0],rowSizeOnGPU,rowSizeOnGPU,nrOfEquations,hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(bOnGPU,b,rowSizeOnGPU,hipMemcpyHostToDevice));

	cudaCheckReturn(hipMemset(xOnGPU,0,rowSizeOnGPU));

	while (++iters<MAX_ITERATIONS && jacobiIterate(cOnGPU,bOnGPU,xOnGPU,deltaXonGPU,nrOfEquations,pitchLength) > 0); // Iterate until convergence

	if (iters==MAX_ITERATIONS) {std::cerr << "Jacobi method did not converge" << std::endl;return -1;}

	cudaCheckReturn(hipMemcpy(x,xOnGPU,rowSizeOnGPU,hipMemcpyDeviceToHost));

	cudaCheckReturn(hipFree(cOnGPU));
	cudaCheckReturn(hipFree(bOnGPU));
	cudaCheckReturn(hipFree(xOnGPU));
	cudaCheckReturn(hipFree(deltaXonGPU));

	// Associate each variable string with its value:
	std::vector<std::pair<std::string,int>> varPairs;
	varPairs.reserve(nrOfEquations);
	for (int i=0;i<nrOfEquations;i++) {
		varPairs.push_back(std::make_pair(variableList[i],x[i]));
	}

	// Sort the strings:
	sort( varPairs.begin(), varPairs.end());

	// Print the result:
	for (int i=0;i<nrOfEquations;i++) {
		std::cout << varPairs[i].first << " = " << varPairs[i].second << std::endl;
	}

	delete[] C[0];
	delete[] C;
	delete[] b;
	delete[] x;
}
